#include "hip/hip_runtime.h"
__global__ void perceptron(float *trainingdata, float *weights, int *n){
	const int i = threadIdx.x;
	for(int j = 0; j < n; j++){
		float result = dot(trainingdata[i][0],weights)
	}
}

__device__ float dot(float *a, float *b, int *n){
	float out = 0.0;
	for(int i = 0; i<n; i++){
		out += a[i] * b[i];
	}
	return out;
}